#include "hip/hip_runtime.h"
#include "cu_deform.h"

__host__ void host_deform(float *d_img1, float *d_img, int nx, int ny, int nz, float volume, float flow, float *alpha_x, float *alpha_y, float *alpha_z, float *beta_x, float *beta_y, float *beta_z)
{
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y, (nz + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
    float *mx, *my, *mz;
    hipMalloc((void**)&mx, nx * ny * nz * sizeof(float));
    hipMalloc((void**)&my, nx * ny * nz * sizeof(float));
    hipMalloc((void**)&mz, nx * ny * nz * sizeof(float));
    kernel_forwardDVF<<<gridSize, blockSize>>>(mx, my, mz, alpha_x, alpha_y, alpha_z, beta_x, beta_y, beta_z, volume, flow, nx, ny, nz);
    hipDeviceSynchronize();
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipPitchedPtr dp_img = make_hipPitchedPtr((void*) d_img, nx * sizeof(float), nx, ny);
    hipMemcpy3DParms copyParams = {0};
    struct hipExtent extent_img = make_hipExtent(nx, ny, nz);
    copyParams.extent = extent_img;
    copyParams.kind = hipMemcpyDeviceToDevice;
    copyParams.srcPtr = dp_img;
    hipArray *array_img;
    hipMalloc3DArray(&array_img, &channelDesc, extent_img);
    copyParams.dstArray = array_img;
    hipMemcpy3D(&copyParams);   

    hipResourceDesc resDesc;
    hipTextureDesc texDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    resDesc.res.array.array = array_img;
    hipTextureObject_t tex_img = 0;
    hipCreateTextureObject(&tex_img, &resDesc, &texDesc, NULL);
    kernel_deformation<<<gridSize, blockSize>>>(d_img1, tex_img, mx, my, mz, nx, ny, nz);
    hipDeviceSynchronize();
    hipFree(mx);   
    hipFree(my);   
    hipFree(mz);   
    hipDestroyTextureObject(tex_img);
    hipFreeArray(array_img);

}

__host__ void host_deform_invert(float *d_img1, float *d_img, int nx, int ny, int nz, float volume, float flow, float *alpha_x, float *alpha_y, float *alpha_z, float *beta_x, float *beta_y, float *beta_z)
{
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y, (nz + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
    float *mx, *my, *mz;
    hipMalloc((void**)&mx, nx * ny * nz * sizeof(float));
    hipMalloc((void**)&my, nx * ny * nz * sizeof(float));
    hipMalloc((void**)&mz, nx * ny * nz * sizeof(float));
    kernel_forwardDVF<<<gridSize, blockSize>>>(mx, my, mz, alpha_x, alpha_y, alpha_z, beta_x, beta_y, beta_z, volume, flow, nx, ny, nz);
    hipDeviceSynchronize();

    float *mx2, *my2, *mz2;
    hipMalloc((void**)&mx2, nx * ny * nz * sizeof(float));
    hipMalloc((void**)&my2, nx * ny * nz * sizeof(float));
    hipMalloc((void**)&mz2, nx * ny * nz * sizeof(float));
    
    host_invert(mx2, my2, mz2, mx, my, mz, nx, ny, nz);
    
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipPitchedPtr dp_img = make_hipPitchedPtr((void*) d_img, nx * sizeof(float), nx, ny);
    hipMemcpy3DParms copyParams = {0};
    struct hipExtent extent_img = make_hipExtent(nx, ny, nz);
    copyParams.extent = extent_img;
    copyParams.kind = hipMemcpyDeviceToDevice;
    copyParams.srcPtr = dp_img;
    hipArray *array_img;
    hipMalloc3DArray(&array_img, &channelDesc, extent_img);
    copyParams.dstArray = array_img;
    hipMemcpy3D(&copyParams);   

    hipResourceDesc resDesc;
    hipTextureDesc texDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    resDesc.res.array.array = array_img;
    hipTextureObject_t tex_img = 0;
    hipCreateTextureObject(&tex_img, &resDesc, &texDesc, NULL);
    kernel_deformation<<<gridSize, blockSize>>>(d_img1, tex_img, mx2, my2, mz2, nx, ny, nz);
    hipDeviceSynchronize();
    hipFree(mx);   
    hipFree(my);   
    hipFree(mz); 
    hipFree(mx2);   
    hipFree(my2);   
    hipFree(mz2);   
    hipDestroyTextureObject(tex_img);
    hipFreeArray(array_img);

}

__host__ void host_invert(float *mx2, float *my2, float *mz2, float *mx, float *my, float *mz, int nx, int ny, int nz)
{
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y, (nz + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipPitchedPtr dp_mx = make_hipPitchedPtr((void*) mx, nx * sizeof(float), nx, ny);
    hipPitchedPtr dp_my = make_hipPitchedPtr((void*) my, nx * sizeof(float), nx, ny);
    hipPitchedPtr dp_mz = make_hipPitchedPtr((void*) mz, nx * sizeof(float), nx, ny);

    hipMemcpy3DParms copyParams = {0};
    struct hipExtent extent = make_hipExtent(nx, ny, nz);
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyDeviceToDevice;
    
    copyParams.srcPtr = dp_mx;
    hipArray *array_mx;
    hipMalloc3DArray(&array_mx, &channelDesc, extent);
    copyParams.dstArray = array_mx;
    hipMemcpy3D(&copyParams);   

    copyParams.srcPtr = dp_my;
    hipArray *array_my;
    hipMalloc3DArray(&array_my, &channelDesc, extent);
    copyParams.dstArray = array_my;
    hipMemcpy3D(&copyParams);   

    copyParams.srcPtr = dp_mz;
    hipArray *array_mz;
    hipMalloc3DArray(&array_mz, &channelDesc, extent);
    copyParams.dstArray = array_mz;
    hipMemcpy3D(&copyParams);   


    hipResourceDesc resDesc;
    hipTextureDesc texDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    resDesc.res.array.array = array_mx;
    hipTextureObject_t tex_mx = 0;
    hipCreateTextureObject(&tex_mx, &resDesc, &texDesc, NULL);

    resDesc.res.array.array = array_my;
    hipTextureObject_t tex_my = 0;
    hipCreateTextureObject(&tex_my, &resDesc, &texDesc, NULL);

    resDesc.res.array.array = array_mz;
    hipTextureObject_t tex_mz = 0;
    hipCreateTextureObject(&tex_mz, &resDesc, &texDesc, NULL);

    kernel_invert<<<gridSize, blockSize>>>(mx2, my2, mz2, tex_mx, tex_my, tex_mz, nx, ny, nz);
    hipDeviceSynchronize();
    hipDestroyTextureObject(tex_mx);
    hipFreeArray(array_mx);
    hipDestroyTextureObject(tex_my);
    hipFreeArray(array_my);
    hipDestroyTextureObject(tex_mz);
    hipFreeArray(array_mz);
}
__global__ void kernel_invert(float *mx2, float *my2, float *mz2, hipTextureObject_t tex_mx, hipTextureObject_t tex_my, hipTextureObject_t tex_mz, int nx, int ny, int nz)
{
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    int iz = BLOCKSIZE_Z * blockIdx.z + threadIdx.z;
    if (ix >= nx || iy >= ny || iz >= nz)
        return;
    int id = ix + iy * nx + iz * nx * ny;
    float x = 0, y = 0, z = 0;
    for (int iter = 0; iter < 10; iter ++){
        x = - tex3D<float>(tex_mx, (x + ix + 0.5f), (y + iy + 0.5f), (z + iz + 0.5f));
        y = - tex3D<float>(tex_my, (x + ix + 0.5f), (y + iy + 0.5f), (z + iz + 0.5f));
        z = - tex3D<float>(tex_mz, (x + ix + 0.5f), (y + iy + 0.5f), (z + iz + 0.5f));
    }
    mx2[id] = x;
    my2[id] = y;
    mz2[id] = z;
}

__host__ void host_deform2(float *d_img1, float *d_img, int nx, int ny, int nz, float volume, float flow, float *alpha_x, float *alpha_y, float *alpha_z, float *beta_x, float *beta_y, float *beta_z)
{
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y, (nz + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
    float *mx, *my, *mz;
    hipMalloc((void**)&mx, nx * ny * nz * sizeof(float));
    hipMalloc((void**)&my, nx * ny * nz * sizeof(float));
    hipMalloc((void**)&mz, nx * ny * nz * sizeof(float));
    kernel_forwardDVF<<<gridSize, blockSize>>>(mx, my, mz, alpha_x, alpha_y, alpha_z, beta_x, beta_y, beta_z, volume, flow, nx, ny, nz);
    hipDeviceSynchronize();
    kernel_deformation2<<<gridSize, blockSize>>>(d_img1, d_img, mx, my, mz, nx, ny, nz);
    hipDeviceSynchronize();
    hipFree(mx);
    hipFree(my);
    hipFree(mz);
}

__global__ void kernel_forwardDVF(float *mx, float *my, float *mz, float *alpha_x, float *alpha_y, float *alpha_z, float *beta_x, float *beta_y, float *beta_z, float volume, float flow, int nx, int ny, int nz)
{
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    int iz = BLOCKSIZE_Z * blockIdx.z + threadIdx.z;
    if (ix >= nx || iy >= ny || iz >= nz)
        return;
    int id = ix + iy * nx + iz * nx * ny;    
    mx[id] = alpha_x[id] * volume + beta_x[id] * flow;
    my[id] = alpha_y[id] * volume + beta_y[id] * flow;
    mz[id] = alpha_z[id] * volume + beta_z[id] * flow;
}

__global__ void kernel_deformation(float *img1, hipTextureObject_t tex_img, float *mx, float *my, float *mz, int nx, int ny, int nz){
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    int iz = BLOCKSIZE_Z * blockIdx.z + threadIdx.z;
    if (ix >= nx || iy >= ny || iz >= nz)
        return;
    int id = iy + ix * ny + iz * nx * ny;
    float xi = iy + 1.0f + my[id];
    float yi = ix + 1.0f + mx[id];
    float zi = iz + 1.0f + mz[id];
    img1[id] = tex3D<float>(tex_img, xi - 0.5f, yi - 0.5f, zi - 0.5f);
}

__global__ void kernel_deformation2(float *img1, float *img, float *mx, float *my, float *mz, int nx, int ny, int nz){
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    int iz = BLOCKSIZE_Z * blockIdx.z + threadIdx.z;
    if (ix >= nx || iy >= ny || iz >= nz)
        return;
    int id = iy + ix * ny + iz * nx * ny;
    int id2 = ix + iy * nx + iz * nx * ny;

    float dx, dy, dz;
    if (ix == nx - 1)
        dx = 0;
    else
        dx = img[id2 + 1] - img[id2];
        
    if (iy == ny - 1)
        dy = 0;
    else
        dy = img[id2 + nx] - img[id2];

    if (iz == nz - 1)
        dz = 0;
    else
        dz = img[id2 + nx * ny] - img[id2];
    img1[id2] = img[id2] + dy * mx[id2] + dx * my[id2] + dz * mz[id2];
}